#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <stdlib.h>

#include "common.h"
#include "cuda_common.cuh"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

// for initializing arrays
void initialize(int* input, const int array_size,
	INIT_PARAM PARAM, int x)
{
	if (PARAM == INIT_ONE)
	{
		for (int i = 0; i < array_size; i++)
		{
			input[i] = 1;
		}
	}
	else if (PARAM == INIT_ONE_TO_TEN)
	{
		for (int i = 0; i < array_size; i++)
		{
			input[i] = i % 10;
		}
	}
	else if (PARAM == INIT_RANDOM)
	{
		time_t t;
		srand((unsigned)time(&t));
		for (int i = 0; i < array_size; i++)
		{
			input[i] = (int)(rand() & 0xFF);
		}
	}
	else if (PARAM == INIT_FOR_SPARSE_METRICS)
	{
		srand(time(NULL));
		int value;
		for (int i = 0; i < array_size; i++)
		{
			value = rand() % 25;
			if (value < 5)
			{
				input[i] = value;
			}
			else
			{
				input[i] = 0;
			}
		}
	}
	else if (PARAM == INIT_0_TO_X)
	{
		srand(time(NULL));
		int value;
		for (int i = 0; i < array_size; i++)
		{
			input[i] = (int)(rand() & 0xFF);
		}
	}
}

int reduction_cpu(int* input, const int size)
{
	int sum = 0;
	for (int i = 0; i < size; i++)
	{
		sum += input[i];
	}
	return sum;
}

void compare_results(int gpu_result, int cpu_result)
{
	printf("GPU result : %d , CPU result : %d \n",
		gpu_result, cpu_result);

	if (gpu_result == cpu_result)
	{
		printf("GPU and CPU results are same \n");
		return;
	}

	printf("GPU and CPU results are different \n");
}

__global__ void reduction_kernel_warp_unrolling(int* int_array,
	int* temp_array, int size)
{
	int tid = threadIdx.x;

	int index = blockDim.x * blockIdx.x + threadIdx.x;

	int* i_data = int_array + blockDim.x * blockIdx.x;

	// In this implementation we would stop the for loop at 64
	// if we continue this furthur, divergence would occur
	for (int offset = blockDim.x / 2; offset >= 64; offset = offset / 2)
	{
		if (tid < offset)
		{
			i_data[tid] += i_data[tid + offset];
		}
		__syncthreads();
	}

	// now the results are in the first 64 threads

	// now we have to do the itertaions from threads 32 to 1

	// since 32 is considered, all the threads in the first warp is fully occupaid.
	// therefore no warp divergence there
	if (tid < 32)
	{	
		// now get data into the volatile pointer

		volatile int* vsmem = i_data;

		// then do all the iterations
		vsmem[tid] += vsmem[tid + 32];

		//the next line will execute once the above execution is over.
		// therefore the results needed for the next line execution would be
		// in the array prior to the next line execution.

		vsmem[tid] += vsmem[tid + 16];
		vsmem[tid] += vsmem[tid + 8];
		vsmem[tid] += vsmem[tid + 4];
		vsmem[tid] += vsmem[tid + 2];
		vsmem[tid] += vsmem[tid + 1];
	}

	if (tid == 0)
	{
		temp_array[blockIdx.x] = i_data[0];
	}
}


int main(int argc, char ** argv)
{
	printf("Running parallel reduction with neighbored pairs improved kernel \n");

	int size = 1 << 27;
	int byte_size = size * sizeof(int);
	int block_size = 128;

	int * h_input, *h_ref;
	h_input = (int*)malloc(byte_size);

 	initialize(h_input, size, INIT_RANDOM);

	int cpu_result = reduction_cpu(h_input, size);

	dim3 block(block_size);
	dim3 grid(size / block.x);

	printf("Kernel launch parameters || grid : %d, block : %d \n", grid.x, block.x);

	int temp_array_byte_size = sizeof(int)* grid.x;

	h_ref = (int*)malloc(temp_array_byte_size);

	int * d_input, *d_temp;
	gpuErrchk(hipMalloc((void**)&d_input, byte_size));
	gpuErrchk(hipMalloc((void**)&d_temp, temp_array_byte_size));

	gpuErrchk(hipMemset(d_temp, 0, temp_array_byte_size));
	gpuErrchk(hipMemcpy(d_input, h_input, byte_size,
		hipMemcpyHostToDevice));

	reduction_kernel_warp_unrolling << < grid, block >> > (d_input, d_temp, size);

	gpuErrchk(hipDeviceSynchronize());
	gpuErrchk(hipMemcpy(h_ref, d_temp, temp_array_byte_size, hipMemcpyDeviceToHost));

	int gpu_result = 0;
	for (int i = 0; i < grid.x; i++)
	{
		gpu_result += h_ref[i];
	}

	compare_results(gpu_result, cpu_result);

	gpuErrchk(hipFree(d_input));
	gpuErrchk(hipFree(d_temp));
	free(h_input);
	free(h_ref);

	gpuErrchk(hipDeviceReset());
	return 0;
}