#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "cuda_common.cuh"

#include <stdlib.h>
#include <time.h>
#include <cstring>

__global__ void sum_array_gpu(int* a, int* b, int* c, int size) {

	// since grids and blocks are 1D
	int gid = threadIdx.x + blockDim.x * blockIdx.x;

	// do the calculation
	if (gid < size) {
		c[gid] = a[gid] + b[gid];
	}

}

// for validity checking
void sum_array_cpu(int* a, int* b, int* c, int size) {

	for (int i = 0;i < size;i++) {
		c[i] = b[i] + a[i];
	}

}

// for validity checking
void compare_arrays(int* a, int* b, int size) {

	for (int i = 0;i < size; i++) {

		if (a[i] != b[i]) {
			printf("Arrays are different i:%d %d %d\n", i, a[i], b[i]);
			return;
		}

	}

	printf("Arrays are the same \n");
}

int main() {

	// size of the array
	int size = 10000;

	// calculate the byte size
	int bytes_size = size * sizeof(int);

	// get the block size
	int block_size = 128;

	// host array pointers and allocation
	int* h_a = (int*)malloc(bytes_size);
	int* h_b = (int*)malloc(bytes_size);

	// to store gpu calculations
	int* gpu_results = (int*)malloc(bytes_size);

	// randomly initialize pointers
	time_t t;
	srand((unsigned)time(&t));

	// assign values to arrays
	for (int i = 0;i < size;i++) {
		h_a[i] = (int)(rand() & 0xff);
		//h_b[i] = (int)(rand() & 0xff);
	}

	for (int i = 0;i < size;i++) {
		//h_a[i] = (int)(rand() & 0xff);
		h_b[i] = (int)(rand() & 0xff);
	}

	// set the gpu results to 0 initially
	memset(gpu_results, 0, bytes_size);

	// device pointers
	int* d_a, * d_b, * d_c;

	// allocate memory on the device
	gpuErrchk(hipMalloc((int**)&d_a, bytes_size));
	gpuErrchk(hipMalloc((int**)&d_b, bytes_size));
	gpuErrchk(hipMalloc((int**)&d_c, bytes_size));

	// copy the h_a and h_b pointers to the device
	gpuErrchk(hipMemcpy(d_a, h_a, bytes_size, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_b, h_b, bytes_size, hipMemcpyHostToDevice));

	// launching the grid
	dim3 block(block_size);

	// we should add one to make sure that there are more blocks than elements
	dim3 grid(size / block.x + 1);

	// launching the kernal
	sum_array_gpu << <grid, block >> > (d_a, d_b, d_c, size);

	// wait until the kernal execution is finished
	gpuErrchk(hipDeviceSynchronize());

	// copy the result back to the host
	gpuErrchk(hipMemcpy(gpu_results, d_c, bytes_size, hipMemcpyDeviceToHost));

	// we dont have a way to confirm if the gpu implementation is correct or not
	// because the array is very large therefore can not print and check each value
	// therefore we need to check the gpu result with the cpu result
	// this is validity checking

	// create a new array for this
	int* h_c = (int*)malloc(bytes_size);

	// sum up using the cpu
	sum_array_cpu(h_a, h_b, h_c, size);

	// now we need to compare two arrays
	// but since that function will come in many places
	// it is ideal to put that function is a header file
	// calling the function from the header file
	compare_arrays(gpu_results, h_c, size);

	// reclaim the memory
	gpuErrchk(hipFree(d_c));
	gpuErrchk(hipFree(d_a));
	gpuErrchk(hipFree(d_b));

	// reclainming the hsot memory
	free(h_a);
	free(h_b);
	free(gpu_results);

	// reset the device
	gpuErrchk(hipDeviceReset());
	return 0;
}



