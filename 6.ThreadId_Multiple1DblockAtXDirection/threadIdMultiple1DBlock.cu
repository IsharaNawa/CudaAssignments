#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void print_value_using_unique_threadId(int * input){

    // get the threadId
    int tid = threadIdx.x;

    // calculate the block offset
    int block_offset = blockIdx.x * blockDim.x;

    // global idx
    int gid = tid + block_offset;

    // print the value
    printf("blockIdx.x: %d , threadIdx.x: %d , globalId: %d value: %d\n",blockIdx.x,threadIdx.x,gid,input[gid]);

}

int main(){


    // define array size 
    int size = 16;

    // calculate the number of bytes for the array
    int array_byte_size = sizeof(int) * size;

    // get a new array
    int h_data[] = {23,9,4,55,65,12,1,33,12,13,56,7,88,45,68,90};

    // print digits
    for(int i=0;i<size;i++){
        printf("%d ",h_data[i]);
    }

    // define an array for data in device
    int & d_data;

    // get space for data inside the device
    hipMalloc((void**)&d_data,array_byte_size);

    // copy the values from host to device
    hipMemcpy(d_data,h_data,array_byte_size,hipMemcpyHostToDevice);
    
    // now define the block size
    dim3 block(4);

    // now define the gird
    dim3 grid(2);

    // now call the kernal
    print_value_using_unique_threadId <<< grid,block >>> (d_data);

    // now synchroize the code
    hipDeviceSynchronize();

    // reset the device
    hipDeviceReset();

    return 0;
}